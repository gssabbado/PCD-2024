#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h> // Para gettimeofday

#include <hip/hip_runtime.h>

#define N 2000
#define T 1000
#define D 0.1
#define DELTA_T 0.01
#define DELTA_X 1.0

__global__ void diff_eq_kernel(double *C, double *C_new, int Num)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < Num - 1 && j > 0 && j < Num - 1) {
        int idx = i * Num + j;
        C_new[idx] = C[idx] + D * DELTA_T *
                    ((C[(i+1)*Num+j] + C[(i-1)*Num+j] + C[i*Num+(j+1)] + C[i*Num+(j-1)] - 4 * C[idx]) /
                     (DELTA_X * DELTA_X));
    }
}

int main()
{
    double *h_C;
    h_C = (double *)malloc(N * N * sizeof(double));

    if (h_C == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        return 1;
    }

    for (int i = 0; i < N * N; i++) {
        h_C[i] = 0.0;
    }
    h_C[(N/2) * N + (N/2)] = 1.0;

    double *d_C, *d_C_new;
    hipMalloc((void**)&d_C, N * N * sizeof(double));
    hipMalloc((void**)&d_C_new, N * N * sizeof(double));

    hipMemcpy(d_C, h_C, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimBlock(4, 2); // Exemplo de dimensões
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    // Medição de tempo
    struct timeval start, end;
    gettimeofday(&start, NULL);

    for (int t = 0; t < T; t++) {
        diff_eq_kernel<<<dimGrid, dimBlock>>>(d_C, d_C_new, N);

        // Sincronizar para garantir que todos os threads terminaram
        hipDeviceSynchronize();

        // Trocar ponteiros para a próxima iteração
        double* temp = d_C;
        d_C = d_C_new;
        d_C_new = temp;
    }

    gettimeofday(&end, NULL);

    hipMemcpy(h_C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);

    printf("Concentração final no centro: %f\n", h_C[(N/2) * N + (N/2)]);

    // Calcular o tempo decorrido
    double elapsedTime = (end.tv_sec - start.tv_sec) * 1000.0; // Segundos para milissegundos
    elapsedTime += (end.tv_usec - start.tv_usec) / 1000.0; // Microssegundos para milissegundos
    printf("Tempo total de execução: %.2f ms\n", elapsedTime);

    hipFree(d_C);
    hipFree(d_C_new);
    free(h_C);

    return 0;
}
